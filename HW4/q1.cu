
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <cstdlib>
#include <math.h>

__global__ void minA_kernel(int *A, int *min, int *lock, int size){
  
  int index  = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = gridDim.x+blockDim.x;
  int offset = 0;
  __shared__ int cache[32];

  int temp = 1000000;
  while(index + offset < size)
  {
    if(temp > A[index+offset])
      temp = A[index+offset];
    offset += stride;
  }
  cache[threadIdx.x] = temp;
 
   __syncthreads();
  
  int i = blockDim.x/2;
  while(i!=0){
    if(threadIdx.x < i){  
      if(cache[threadIdx.x] > cache[threadIdx.x+i]);
        cache[threadIdx.x] = cache[threadIdx.x+i];
    }
    __syncthreads();
    i /=2;
  }
  
  if(threadIdx.x == 0){
    while(atomicCAS(lock,0,1) != 0){}
    if(*min > cache[0]);
      *min = cache[0];
    atomicExch(lock, 0);
  } 
}

__global__ void lastDigit_kernel(int* A, int* B, int size)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = gridDim.x * blockDim.x;
  int offset = 0;
  while(index  + offset < size)
  {
     B[index + offset] = A[index+offset]%10;
     offset += stride;
  }

}

int main(int argc, char* argv[]){
 using namespace std;
 if(argc>1)
  {
  ifstream in(argv[1]);
  string line;
  getline(in, line);
  stringstream ss(line);
  string field;
  vector<int> v;
  string::size_type sz;
  while(getline(ss,field,','))
  { 
    int n = field.length();
    char char_array[n+1];
    strcpy(char_array, field.c_str()); 
    int var = atoi(char_array);
  
    v.push_back(var);
  }
 
 int* A = &v[0];
 int size = v.size();
 int *d_A;
 int *h_min;
 int *d_min;
 int *d_lock;
 int* B;
 int* d_B;
 int adjustedSize = 1;
 int* adjA;
 
 while(adjustedSize < size)
   adjustedSize *= 2;
 

 h_min = (int*)malloc(sizeof(int));
 hipMalloc((void**)&d_A, adjustedSize*sizeof(int));
 hipMalloc((void**)&d_min, sizeof(int));
 hipMalloc((void**)&d_lock, sizeof(int));
 hipMemset(d_min, 100000, sizeof(int));
 hipMemset(d_lock, 0, sizeof(int));
 adjA = (int*)malloc(adjustedSize*sizeof(int));
 memcpy(adjA, A, adjustedSize*sizeof(int));

 B = (int*)malloc(size*sizeof(int));
 hipMalloc((void**)&d_B,size*sizeof(int));
 
 for(int i = size;i<adjustedSize;i++){
    adjA[i] = 1000;
 }
 hipMemcpy(d_A, adjA, size*sizeof(int), hipMemcpyHostToDevice);
     
  dim3 gridSize = 32;
  dim3 blockSize = 32;
  minA_kernel<<< gridSize, blockSize>>>(d_A, d_min, d_lock, adjustedSize);
  
  hipMemcpy(h_min, d_min, sizeof(int), hipMemcpyDeviceToHost);
  ofstream myfile;
  myfile.open("q1a.txt");
  
   
  myfile<<"Min is"<<*h_min;
  
  int min = 100000;
  for(int i=0;i<size;i++)
  {
    if(A[i] <  min){
      min = A[i];
    }
  }

  myfile<<"Non Parallel Min is " <<min;
  myfile.close();
  myfile.open("q1b.txt");

  dim3 gridSize2 = size/32 +1;
  dim3 blockSize2 = 32;
  lastDigit_kernel<<<gridSize2, blockSize2>>>(d_A,d_B,size);
  hipMemcpy(B,d_B,size*sizeof(int),hipMemcpyDeviceToHost);
  for(int i=0;i<size-1;i++)
  {
    myfile<<B[i]<<", ";
  }
  myfile<<B[size-1];
  myfile.close();

  /*free(A);
  free(h_min);
  free(d_A);
  free(d_min);
  free(d_lock);*/
 }
 else{
   cout<<"No Arguments";
 }
 return 0;
}

