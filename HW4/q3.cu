
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <cstdlib>
#include <math.h>

__global__ void countodd_kernel(int *A, int *count, int *values, int size){
  
  int index  = threadIdx.x + blockIdx.x*(size/gridDim.x);
  int stride = 32;
  int offset = 0;
  __shared__ int cache[32];
  cache[threadIdx.x] = 0;

  int temp = 0;
  while(index + offset <(blockIdx.x+1)*(size/gridDim.x))
  {
    if(A[index + offset]%2==1)
      temp+=1;
    offset += stride;
  }
  cache[threadIdx.x] = temp;
 
   __syncthreads();
  
  int i = blockDim.x/2;
  while(i!=0){
    if(threadIdx.x < i){  
      cache[threadIdx.x] += cache[threadIdx.x + i];
    }
    __syncthreads();
    i /= 2; 
  }

  if(threadIdx.x==0){
    atomicAdd(count,cache[0]);
    values[blockIdx.x] = cache[0];
  }
}

__global__ void getodd_kernel(int *A, int *D, int sizeA, int *sizeD){
   
   int index = threadIdx.x + blockIdx.x*(sizeA/gridDim.x);
   int stride = 1;
   int offset = 0;
   int count = 0;
   int startIndex = 0;
   for(int i=0;i<blockIdx.x;i++)
   {
     startIndex+=sizeD[i];
   }
   while(count<sizeD[blockIdx.x])
   {
     if(A[index+offset]%2==1)
     {
       D[startIndex+count] = A[index+offset];
       count++;  
     }
     offset += stride;   
   }
  
}

int main(int argc, char* argv[]){
 using namespace std;
 if(argc>1)
  {
  ifstream in(argv[1]);
  string line;
  getline(in, line);
  stringstream ss(line);
  string field;
  vector<int> v;
  string::size_type sz;
  while(getline(ss,field,','))
  { 
    int n = field.length();
    char char_array[n+1];
    strcpy(char_array, field.c_str()); 
    int var = atoi(char_array);
  
    v.push_back(var);
  }
 
 int* A = &v[0];
 int size = v.size();
 int* d_A;
 int *d_count;
 int *h_count;
 int *d_lock;
 int adjustedSize = 1;
 int* adjA;
 int* d_values;
 int* h_values;
 
 while(adjustedSize < size)
   adjustedSize *= 2;
 cout<<adjustedSize<<endl;

 h_values = (int*)malloc(32*sizeof(int));
 hipMalloc((void**)&d_values, 32*sizeof(int));
 h_count = (int*)malloc(sizeof(int));
 hipMalloc((void**)&d_A, adjustedSize*sizeof(int));
 hipMalloc((void**)&d_count, sizeof(int));
 hipMalloc((void**)&d_lock, sizeof(int));
 hipMemset(d_count, 0, sizeof(int));
 hipMemset(d_lock, 0, sizeof(int));
 hipMemset(d_values, 0, 32*sizeof(int));
 adjA = (int*)malloc(adjustedSize*sizeof(int));
 memcpy(adjA, A, size*sizeof(int));

 // B = (int*)malloc(size*sizeof(int));
// cudaMalloc((void**)&d_B,size*sizeof(int));
 
 for(int i = size;i<adjustedSize;i++){
    adjA[i] = 8;
 }
 hipMemcpy(d_A, adjA, adjustedSize*sizeof(int), hipMemcpyHostToDevice);
     
  dim3 gridSize = 32;
  dim3 blockSize = 32;
  countodd_kernel<<< 32, 32>>>(d_A, d_count, d_values, adjustedSize);
  
  hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_values, d_values, 32*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(d_values, h_values, 32*sizeof(int), hipMemcpyHostToDevice);  

  for(int i=0;i<32;i++)
     cout<<i<<": "<<h_values[i]<<endl;
 /* int* h_D[32];
  int* d_D[32];
  for(int i=0;i<32;i++)
     h_D[i] = (int*)malloc(h_values[i]*sizeof(int));
  for(int i=0;i<32;i++)
     cudaMalloc((void**)&d_D[i],h_values[i]*sizeof(int));

  getodd_kernel<<<32,1>>>(d_A, d_D, adjustedSize, d_values);
  for(int i=0;i<32;i++)
    cudaMemcpy(h_D[i],d_D[i],h_values[i]*sizeof(int),cudaMemcpyDeviceToHost);
  for(int i=0;i<32;i++)
    for(int j=0;j<h_values[i];j++)
      cout<<h_D[i][j]<<", ";*/
  
  ofstream(myfile);
  myfile.open("q3.txt");  

  int* d_D;
  int* h_D;
  h_D = (int*)malloc(*h_count*sizeof(int));
  hipMalloc((void**)&d_D,*h_count*sizeof(int));
  getodd_kernel<<<32,1>>>(d_A, d_D, adjustedSize, d_values);
  
  hipMemcpy(h_D, d_D, *h_count*sizeof(int), hipMemcpyDeviceToHost);
  for(int i=0;i<*h_count-1;i++)
     myfile<<h_D[i]<<", ";
  myfile<<h_D[*h_count-1];
  
      
  cout<<"Count is "<<*h_count;
  
  int count = 0;
  for(int i=0;i<size;i++)
  {
    if(A[i] %2==1){
      count += 1;
    }

  }

  cout<<" Seq Count is "<<count;


  /*free(A);
  free(h_min);
  free(d_A);
  free(d_min);
  free(d_lock);*/
 }
 else{
   cout<<"No Arguments";
 }
 return 0;
}

