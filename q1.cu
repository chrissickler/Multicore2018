
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <cstdlib>
int main(int argc, char* argv[]){
  using namespace std;
 if(argc>1)
  {
  ifstream in(argv[1]);
  string line;
  getline(in, line);
  stringstream ss(line);
  string field;
  vector<int> arr;
  string::size_type sz;
  while(getline(ss,field,','))
  { 
    int n = field.length();
    char char_array[n+1];
    strcpy(char_array, field.c_str()); 
    int var = atoi(char_array);

    arr.push_back(var);
  }
  for(int i =0;i<arr.size();i++)
  {
    cout << arr[i] << " ";
  }
 }
 else
 {
  std::cout << "No arguments." << '\n';
 }
 return 0;
}
