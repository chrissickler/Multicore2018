#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <cstdlib>

#define NUM_BLOCKS 32
#define BLOCK_WIDTH 32

__global__ void countG(int *a, int *b, int n, int *mutex) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int offset = 0;

    while (index + offset < n) {
        int val = a[index+offset]/100;
        
    }
}

__global__ void count(int *a, int *b, int n, int *mutex) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int offset = 0;

    int temp_array[10];
    //temp_array = (int*)malloc(10*sizeof(int));

    int i;
    for ( i=0;i<10;i++) {
        temp_array[i] = 0;
        b[i]= 0;
    }

    __shared__ int cache[BLOCK_WIDTH][10];
    for(int j=0;j<BLOCK_WIDTH;j++) {
        for (int k=0;k<10;k++) {
            cache[j][k] = 0;
        }
    }

    while (index + offset < n) {
        int comb = index + offset;
        int val = a[comb]/100;
        temp_array[val] ++;

        offset += stride;
    }

    for (i=0;i<10;i++){
        cache[threadIdx.x][i] = temp_array[i];
    }

    __syncthreads();

    i = blockDim.x/2;
    while (i!=0) {
        if (threadIdx.x < i) {
            for (int j = 0; j < 10;j++) {
                cache[threadIdx.x][j] += cache[threadIdx.x + i][j];
            }
        }
        __syncthreads();
        i/=2;
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(mutex,0,1) != 0);
        for (int j =0; j < 10; j++) {   
            b[j] += cache[0][j];
        }
        atomicExch(mutex,0);
    }   
    
}

int main(int argc, char* argv[]) {
  using namespace std;
  vector<int> arr;
 if(argc>1) {
  ifstream in(argv[1]);
  string line;
  getline(in, line);
  stringstream ss(line);
  string field;
  //string::size_type sz;
  while(getline(ss,field,',')) { 
    int n = field.length();
    char char_array[n+1];
    strcpy(char_array, field.c_str()); 
    int var = atoi(char_array);

    arr.push_back(var);
  }
  
 } else {
  std::cout << "No arguments." << '\n';
 }

    int* countA = (int*)malloc(10*sizeof(int));
    for (int i = 0; i < 10; i++) {
        countA[i] = 0;
    }
    for(int i = 0; i<10000;i++) {
        int val = arr[i]/100;
        countA[val]++;
    }

 
    std::cout<<"count0: "<<countA[0]<<std::endl;
    std::cout<<"count1: "<<countA[1]<<std::endl;
    std::cout<<"count2: "<<countA[2]<<std::endl;
    std::cout<<"count3: "<<countA[3]<<std::endl;
    std::cout<<"count4: "<<countA[4]<<std::endl;
    std::cout<<"count5: "<<countA[5]<<std::endl;
    std::cout<<"count6: "<<countA[6]<<std::endl;
    std::cout<<"count7: "<<countA[7]<<std::endl;
    std::cout<<"count8: "<<countA[8]<<std::endl;
    std::cout<<"count9: "<<countA[9]<<std::endl;
    
    
    int *d_B;
    int *B;
    int *d_A;
    int *d_mutex;
    int *h_A;
    int *C;
    int *d_C;
    int N = arr.size();

    //allocate memory
    h_A = (int*)malloc(N*sizeof(int));
    B = (int*)malloc(10*sizeof(int));
    C = (int*)malloc(10*sizeof(int));
    hipMalloc((void**)&d_A,N*sizeof(int));
    hipMalloc((void**)&d_B, 10*sizeof(int));
    hipMalloc((void**)&d_mutex,sizeof(int));
    hipMalloc((void**)&d_C,10*sizeof(int));

    ofstream q2af;
    ofstream q2bf;
    ofstream q2cf;
    q2af.open("q2a.txt");
    q2bf.open("q2b.txt");
    q2cf.open("q2c.txt");



    //copy from host to device
    h_A = &arr[0];
    /*
    std::cout<<"0: "<<h_A[0]<<std::endl;
    std::cout<<"1: "<<h_A[1]<<std::endl;
    std::cout<<"2: "<<h_A[2]<<std::endl;
    std::cout<<"3: "<<h_A[3]<<std::endl;
    std::cout<<"4: "<<h_A[4]<<std::endl;
    std::cout<<"5: "<<h_A[5]<<std::endl;
    std::cout<<"6: "<<h_A[6]<<std::endl;
    std::cout<<"7: "<<h_A[7]<<std::endl;
    std::cout<<"8: "<<h_A[8]<<std::endl;
    std::cout<<"9: "<<h_A[9]<<std::endl;*/
    hipMemcpy(d_A,h_A,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemset(d_mutex, 0, sizeof(int));

    
    
    /*for (int i = 0; i < 10; i++) {
        B[i] = 0;
    }
    hipMemcpy(d_B,B,10*sizeof(int),hipMemcpyHostToDevice);*/

    count<<<NUM_BLOCKS,BLOCK_WIDTH>>>(d_A,d_B,N,d_mutex);

    hipMemcpy(B,d_B,10*sizeof(int),hipMemcpyDeviceToHost);

    //report results
    /*
    std::cout<<"0-99: "<<B[0]<<std::endl;
    std::cout<<"100-199: "<<B[1]<<std::endl;
    std::cout<<"200-299: "<<B[2]<<std::endl;
    std::cout<<"300-399: "<<B[3]<<std::endl;
    std::cout<<"400-499: "<<B[4]<<std::endl;
    std::cout<<"500-599: "<<B[5]<<std::endl;
    std::cout<<"600-699: "<<B[6]<<std::endl;
    std::cout<<"700-799: "<<B[7]<<std::endl;
    std::cout<<"800-899: "<<B[8]<<std::endl;
    std::cout<<"900-999: "<<B[9]<<std::endl;*/

    q2bf << B[0];
    for (int k = 1; k < 10; k++) {
        q2bf << ", " << B[k];
    }

    hipMemcpy(d_B,B,10*sizeof(int),hipMemcpyHostToDevice);
    hipMemset(d_mutex, 0, sizeof(int));

    count<<<4,4>>>(d_B,d_C,10,d_mutex);

    hipMemcpy(C,d_C,10*sizeof(int),hipMemcpyDeviceToHost);

    q2cf << C[0];
    for (int k = 1; k < 10; k++) {
        q2cf << ", " << C[k];
    }

    /*
    std::cout<<"0-99: "<<C[0]<<std::endl;
    std::cout<<"100-199: "<<C[1]<<std::endl;
    std::cout<<"200-299: "<<C[2]<<std::endl;
    std::cout<<"300-399: "<<C[3]<<std::endl;
    std::cout<<"400-499: "<<C[4]<<std::endl;
    std::cout<<"500-599: "<<C[5]<<std::endl;
    std::cout<<"600-699: "<<C[6]<<std::endl;
    std::cout<<"700-799: "<<C[7]<<std::endl;
    std::cout<<"800-899: "<<C[8]<<std::endl;
    std::cout<<"900-999: "<<C[9]<<std::endl;*/


    free(h_B);
    hipFree(d_B);
    hipFree(d_A);
    hipFree(d_mutex);

    // launch the kernel
    //count<<<NUM_BLOCKS, BLOCK_WIDTH>>>(arr,b);

    // force the printf()s to flush
    //hipDeviceSynchronize();

    //printf("That's all!\n");
    //hipFree(b);
    q2af.close();
    q2bf.close();
    q2cf.close();
 return 0;
}
