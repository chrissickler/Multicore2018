
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <cstdlib>

#define NUM_BLOCKS 32
#define BLOCK_WIDTH 32

__global__ void countG(int *a, int *b, int n, int *mutex) {
    int index = 0;
    while (index < n) {
        int val = a[index]/100;
        if (val == threadIdx.x) {
            b[val]++;
        }
        index++;
    }
}

__global__ void count(int *a, int *b, int n, int *mutex) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    int offset = 0;

    int temp_array[10];
    //temp_array = (int*)malloc(10*sizeof(int));

    int i;
    for ( i=0;i<10;i++) {
        temp_array[i] = 0;
        b[i]= 0;
    }

    __shared__ int cache[BLOCK_WIDTH][10];
    for(int j=0;j<BLOCK_WIDTH;j++) {
        for (int k=0;k<10;k++) {
            cache[j][k] = 0;
        }
    }

    while (index + offset < n) {
        int comb = index + offset;
        int val = a[comb]/100;
        temp_array[val] ++;

        offset += stride;
    }

    for (i=0;i<10;i++){
        cache[threadIdx.x][i] = temp_array[i];
    }

    __syncthreads();

    i = blockDim.x/2;
    while (i!=0) {
        if (threadIdx.x < i) {
            for (int j = 0; j < 10;j++) {
                cache[threadIdx.x][j] += cache[threadIdx.x + i][j];
            }
        }
        __syncthreads();
        i/=2;
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(mutex,0,1) != 0);
        for (int j =0; j < 10; j++) {   
            b[j] += cache[0][j];
        }
        atomicExch(mutex,0);
    }   
    
}

int main(int argc, char* argv[]) {
  using namespace std;
  vector<int> arr;
 if(argc>1) {
  ifstream in(argv[1]);
  string line;
  getline(in, line);
  stringstream ss(line);
  string field;
  //string::size_type sz;
  while(getline(ss,field,',')) { 
    int n = field.length();
    char char_array[n+1];
    strcpy(char_array, field.c_str()); 
    int var = atoi(char_array);

    arr.push_back(var);
  }
  
 } else {
  std::cout << "No arguments." << '\n';
 }

    int* countA = (int*)malloc(10*sizeof(int));
    for (int i = 0; i < 10; i++) {
        countA[i] = 0;
    }
    for(int i = 0; i<10000;i++) {
        int val = arr[i]/100;
        countA[val]++;
    }

    
    
    int *d_B;
    int *B;
    int *d_A;
    int *d_mutex;
    int *h_A;
    int *C;
    int *d_C;
    int N = arr.size();

    //allocate memory
    h_A = (int*)malloc(N*sizeof(int));
    B = (int*)malloc(10*sizeof(int));
    C = (int*)malloc(10*sizeof(int));
    hipMalloc((void**)&d_A,N*sizeof(int));
    hipMalloc((void**)&d_B, 10*sizeof(int));
    hipMalloc((void**)&d_mutex,sizeof(int));
    hipMalloc((void**)&d_C,10*sizeof(int));

    ofstream q2af;
    ofstream q2bf;
    ofstream q2cf;
    q2af.open("q2a.txt");
    q2bf.open("q2b.txt");
    q2cf.open("q2c.txt");



    //copy from host to device
    h_A = &arr[0];

    hipMemcpy(d_A,h_A,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemset(d_mutex, 0, sizeof(int));

    
    
    /*for (int i = 0; i < 10; i++) {
        B[i] = 0;
    }
    cudaMemcpy(d_B,B,10*sizeof(int),cudaMemcpyHostToDevice);*/

    countG<<<1,10>>>(d_A,d_B,N,d_mutex);

    hipMemcpy(B,d_B,10*sizeof(int),hipMemcpyDeviceToHost);

    q2af << B[0];
    for (int k = 1; k < 10; k++) {
        q2af << ", " << B[k];
    }



    count<<<NUM_BLOCKS,BLOCK_WIDTH>>>(d_A,d_B,N,d_mutex);

    hipMemcpy(B,d_B,10*sizeof(int),hipMemcpyDeviceToHost);

    //report results
    

    q2bf << B[0];
    for (int k = 1; k < 10; k++) {
        q2bf << ", " << B[k];
    }

    hipMemcpy(d_B,B,10*sizeof(int),hipMemcpyHostToDevice);
    hipMemset(d_mutex, 0, sizeof(int));

    count<<<4,4>>>(d_B,d_C,10,d_mutex);

    hipMemcpy(C,d_C,10*sizeof(int),hipMemcpyDeviceToHost);

    q2cf << C[0];
    for (int k = 1; k < 10; k++) {
        q2cf << ", " << C[k];
    }



    //free(h_B);
    free(h_A);
    free(B);
    free(C);
    hipFree(d_B);
    hipFree(d_A);
    hipFree(d_C);
    hipFree(d_mutex);

    // force the printf()s to flush
    //cudaDeviceSynchronize();

    //printf("That's all!\n");
    //cudaFree(b);
    q2af.close();
    q2bf.close();
    q2cf.close();
 return 0;
}
